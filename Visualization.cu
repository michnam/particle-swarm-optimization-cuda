#include "hip/hip_runtime.h"
#include "Visualization.cuh"
#include <iostream>

#define TARGET_HEIGHT 1000
#define TARGET_WIDTH 1500

Visualization::Visualization(float minX, float maxX, float minY, float maxY, float (*f)(float, float),
                             std::vector<std::vector<Particle>> posList) : minX(minX),
                                                                           maxX(maxX),
                                                                           minY(minY),
                                                                           maxY(maxY) {
    int iteration = 0;
    int frame = 0;

    setupScale(minX, maxX, minY, maxY);

    sf::RenderWindow window(sf::VideoMode(width, height), "Particle swarm Optimisation - visualization",
                            sf::Style::Titlebar | sf::Style::Close);
    window.setFramerateLimit(60);
    float **map;
    float max;
    float min;
    sf::Image image;
    sf::Texture tex;
    sf::Sprite background;

    calculateFunctionValues(minX, minY, f, map, max, min);
    image.create(width, height, sf::Color(100, 0, 0));
    createMap(minX, minY, f, map, max, min, image);
    tex.loadFromImage(image);
    background.setTexture(tex);


    sf::CircleShape  particle(3, 8);
    particle.setOrigin(3, 3);
    particle.setFillColor(sf::Color(100, 0, 0));

    while (window.isOpen()) {
        handleEvents(window);
        window.clear();
        window.draw(background);


        for (Particle p: posList[iteration]) {
            Position pixel = particleToPixel(p.current_position);
            particle.setPosition(pixel.x, pixel.y);
            window.draw(particle);
        }


        window.display();

        if (frame == 20) {
            frame = 0;
            iteration++;
        }
        if (iteration == posList.size()) iteration = posList.size() - 1;
        frame++;
    }


}

void Visualization::handleEvents(sf::RenderWindow &window) const {
    sf::Event event;
    while (window.pollEvent(event)) {
        if (event.type == sf::Event::Closed)
            window.close();
    }
}

void Visualization::createMap(float minX, float minY, float (*f)(float, float), float **map, float max, float min,
                              sf::Image &image) {
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            int val = (map[i][j] - min) * 255 / (max - min);
            image.setPixel(j, i, sf::Color(val, val, val));
        }
    }
}

void Visualization::calculateFunctionValues(float minX, float minY, float (*f)(float, float), float **&map, float &max,
                                            float &min) {
    map = new float *[height];
    max = f(minX, minY);
    min = f(minX, minY);
    for (int i = 0; i < height; i++)
        map[i] = new float[width];
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            Position pixel(j, i);
            Position p = pixelToParticle(pixel);
            int y = (int) p.y;
            int x = (int) p.x;
            float val = f(x, y);
            map[i][j] = val;
            if (val > max) max = val;
            if (val < min) min = val;
        }
    }
    std::cout << "min: " << min << std::endl;
    std::cout << "max: " << max << std::endl;
}

void Visualization::setupScale(float minX, float maxX, float minY, float maxY) {
    double xScale = TARGET_WIDTH / (maxX - minX);
    double yScale = TARGET_HEIGHT / (maxY - minY);
    std::cout << "minX: " << minX << std::endl;
    std::cout << "maxX: " << maxX << std::endl;
    std::cout << "minY: " << minY << std::endl;
    std::cout << "maxY: " << maxY << std::endl;
    std::cout << "xScale: " << xScale << std::endl;
    std::cout << "yScale: " << yScale << std::endl;

    if (xScale < yScale) {
        scale = xScale;
        width = TARGET_WIDTH;
        height = (maxY - minY) * scale;
    } else {
        scale = yScale;
        height = TARGET_HEIGHT;
        width = (maxX - minX) * scale;
    }
    std::cout << "scale: " << scale << std::endl;
    std::cout << "Creating window - width: " << width << ", height: " << height << std::endl;
}

Position Visualization::particleToPixel(Position particle) {
    //std::cout << (particle.x - minX) * scale << ", " << (particle.y - minY) * scale << std::endl;
    return Position((particle.x - minX) * scale, (particle.y - minY) * scale);
}

Position Visualization::pixelToParticle(Position pixel) {
    return Position(pixel.x / scale + minX, pixel.y / scale + minY);
}

void Position::operator=(const Position &a) {
    x = a.x;
    y = a.y;
}

void Position::operator+=(const Position &a) {
    x = x + a.x;
    y = y + a.y;
}

Position::Position(float x, float y) : x(x), y(y) {}

